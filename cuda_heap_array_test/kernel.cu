#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>

#include <stdio.h>
#include <hip/device_functions.h>
#include <stdlib.h>

__device__ void printarr(int a[1000]) {
    int row = 0;
    for (int i = 0; i < 1000; i++) 
    {
        if (row == 20)
        {
            row = 0;
            printf("\n");
        }
        printf(" %d ", a[i]);
        row++;
    }
}

__device__ bool fill = false;

__global__ void cuda_test()
{
  
    int a[1000];
    if (blockIdx.x == 0)
    {
        for (int i = 0; i < 1000; i++)
        {
            a[i] = 0;
        }
        printarr(a);
        printf("\n\n\n\n");
        fill = true;
    }
    while (!fill) {}
    for (int i = int(256 * blockIdx.x); i < (256 * (int)(blockIdx.x+1));i++)
    {
        a[i] = 5;
    }

    if (blockIdx.x == 0)
    {
        printarr(a);
        printf("\n\n\n\n");
    }

    printf("\nI am cuda number %d:%d", threadIdx.x,blockIdx.x);
}

int main()
{
    cuda_test << <4, 1>> > ();
    hipDeviceSynchronize();
    printf("\nI am the CPU!");
    return 0;

}
