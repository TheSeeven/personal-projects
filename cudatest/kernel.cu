#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <time.h>
#include <stdlib.h> 

__device__ int* ptrarr = 0;
__device__ bool fill=false;
__device__ bool printed = false;
__device__ unsigned int i = 0;
__device__ unsigned int j = 0;
__device__ unsigned int row=0;

__device__ int* generate_arr(const unsigned int x, const unsigned int y)
	{
		int* a = (int*)malloc(x * y * sizeof(int));
		for (i = 0; i < x; i++)
			for (j = 0; j < y; j++)
				*(a + i * y + j) = 0;
		return a;
	}

__device__ void printarr(int* arr, const unsigned int x, const unsigned int y) {
	for (i = 0; i < x; i++) 
	{
		for (j = 0; j < y; j++) 
		{
			if (row == x)
			{
				row = 0;
				printf("\n");
			}
			printf("%d ", *(arr + i * y + j));
			row++;
		}
	}
}


__global__ void array_increment(const unsigned int x,const unsigned int y)
{
	
	if (threadIdx.x == 0)
	{
		ptrarr = generate_arr(x, y);
		printf("\n\nFilling array... \n\n");
		printarr(ptrarr,x,y);
		fill = true;
		printed = true;
	}
	while (!fill) {}
	if (fill && printed)
	{
		for (i = 0; i < x; i++) 
		{
			*(ptrarr + threadIdx.x * x + i) = threadIdx.x;
		}
	}
	if (threadIdx.x == 0)
	{
		printf("\n\narr after increment... \n\n");
		printarr(ptrarr, x, y);
		printf("\n");
	}
}

int main()
{
	int a = 10;
	int b = 20;
	clock_t start, end;
	double t;
	start = clock();
	array_increment << <1, b >> > (a, b);
	//fill_gb<< <1, 1 >> >(1);
	hipDeviceSynchronize();
	end = clock();
	t = ((double)((double)end - (double)start)) / CLOCKS_PER_SEC;
	printf("t = %lf", t);

	printf("\nI am a CPU\n\n\n\n!");

	return 0;
}
